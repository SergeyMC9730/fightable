#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>

// Vector2, 2 components
typedef struct Vector2 {
    float x;                // Vector x component
    float y;                // Vector y component
} Vector2;

// Rectangle, 4 components
typedef struct Rectangle {
    float x;                // Rectangle top-left corner position x
    float y;                // Rectangle top-left corner position y
    float width;            // Rectangle width
    float height;           // Rectangle height
} Rectangle;

__device__ unsigned char CheckCollisionPointRec(Vector2 point, Rectangle rec) {
    if ((point.x >= rec.x) && (point.x < (rec.x + rec.width)) && (point.y >= rec.y) && (point.y < (rec.y + rec.height))) {
        return 1;
    }

    return 0;
}

__global__ void helloFromGPU(Rectangle area, Vector2 *blocks, unsigned char tile_w,unsigned char tile_h, int block_amount, unsigned char *allows) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= block_amount) return;

    Vector2 block = blocks[i];
    allows[i] = CheckCollisionPointRec(Vector2{block.x * tile_w, block.y * tile_h}, area);
}

#include "test.h"

void setup() {
    cudaInitDevice(0, 0, 0);
}

void performGpuAction() {
    constexpr int n = 128;
    Vector2 *host_objects = (Vector2 *)calloc(128, sizeof(Vector2));
    unsigned char *host_allows = (unsigned char *)calloc(128, 1);

    for (int i = 0; i < n; i++) {
        Vector2 *ref = host_objects + i;
        ref->x = i;
        ref->y = i;
    }

    Vector2 *gpu_objects;
    hipMalloc(&gpu_objects, sizeof(Vector2) * n);

    unsigned char *gpu_allows;
    hipMalloc(&gpu_allows, 1 * n);

    Rectangle area = (Rectangle){
        0,
        0,
        160,
        120
    };

    auto start_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(gpu_objects, host_objects, 128 * sizeof(Vector2), hipMemcpyHostToDevice);

    constexpr int threadsPerBlock = 128;
    constexpr int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    helloFromGPU<<<blocksPerGrid, threadsPerBlock>>>(area, gpu_objects, 8, 8, n, gpu_allows);

    hipMemcpy(host_allows, gpu_allows, n, hipMemcpyDeviceToHost);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto time = end_time - start_time;

    for (int i = 0; i < n; i++) {
        printf("obj %f:%f -> %d\n", host_objects[i].x, host_objects[i].x, host_allows[i]);
    }

    free(host_objects);
    free(host_allows);

    hipFree(gpu_allows);
    hipFree(gpu_objects);

    hipDeviceReset();

    printf("\n-----------\ntook %ld ns to execute\n-----------\n",
        time.count()
    );
}