#include <fightable/renderer.h>
#include <fightable/state.h>

void _fPtxInit() {
    TraceLog(LOG_INFO, "Initializing CUDA");

    hipError_t status = cudaInitDevice(0, 0, 0);
    if (status != hipSuccess) {
        TraceLog(LOG_ERROR, "Could not initialize CUDA: %d", (int)status);
    } else {
        __state.can_use_gpu_accel = 1;
    }
}