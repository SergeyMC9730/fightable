#include <fightable/level.h>
#include <fightable/block.h>

void _fLevelReloadCudaCtx(struct flevel *level) {
    if (level->old_block_amount != level->data_size) {
        TraceLog(LOG_INFO, "Updating CUDA context (old size: %d, new size: %d)", level->old_block_amount, level->data_size);

        if (level->host_allow_pipeline) {
            level->host_allow_pipeline = (unsigned char *)realloc(level->host_allow_pipeline, level->data_size);
        } else {
            level->host_allow_pipeline = (unsigned char *)malloc(level->data_size);
        }
        

        if (level->dev_block_pipeline) {
            hipFree(level->dev_block_pipeline);
        }
        hipMalloc(&level->dev_block_pipeline, sizeof(fblock) * level->data_size);
        hipMemcpy(level->dev_block_pipeline, level->objects, sizeof(fblock) * level->data_size, hipMemcpyHostToDevice);

        if (level->dev_allow_pipeline) {
            hipFree(level->dev_allow_pipeline);
        }
        hipMalloc(&level->dev_allow_pipeline, level->data_size);
        
        level->old_block_amount = level->data_size;

        TraceLog(LOG_INFO, "Update complete");
    }
}