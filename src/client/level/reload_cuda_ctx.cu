#include <fightable/level.h>
#include <fightable/block.h>

void _fLevelReloadCudaCtx(struct flevel *level) {
    if (level->old_block_amount != level->data_size) {
        TraceLog(LOG_INFO, "Updating CUDA context (old size: %d, new size: %d)", level->old_block_amount, level->data_size);

        if (level->host_allow_pipeline) {
            level->host_allow_pipeline = (unsigned char *)realloc(level->host_allow_pipeline, level->data_size);
        } else {
            level->host_allow_pipeline = (unsigned char *)malloc(level->data_size);
        }
        if (level->host_block_pipeline) {
            level->host_block_pipeline = (Vector2 *)realloc(level->host_block_pipeline, level->data_size * sizeof(Vector2));
        } else {
            level->host_block_pipeline = (Vector2 *)malloc(level->data_size * sizeof(Vector2));
        }

        for (int i = 0; i < level->data_size; i++) {
            struct fblock block = level->objects[i];
            level->host_block_pipeline[i] = Vector2{(float)block.base.block_x, (float)block.base.block_y};
        }

        if (level->dev_block_pipeline) {
            hipFree(level->dev_block_pipeline);
        }
        hipMalloc(&level->dev_block_pipeline, sizeof(Vector2) * level->data_size);
        hipMemcpy(level->dev_block_pipeline, level->host_block_pipeline, sizeof(Vector2) * level->data_size, hipMemcpyHostToDevice);

        if (level->dev_allow_pipeline) {
            hipFree(level->dev_allow_pipeline);
        }
        hipMalloc(&level->dev_allow_pipeline, level->data_size);

        level->old_block_amount = level->data_size;

        TraceLog(LOG_INFO, "Update complete");
    }
}