#include <fightable/level.h>
#include <fightable/block.h>

void _fLevelDestroyPtx(struct flevel *level) {
    if (!level) return;

    if (level->dev_block_pipeline) hipFree(level->dev_block_pipeline);
    if (level->dev_allow_pipeline) hipFree(level->dev_allow_pipeline);

    level->dev_block_pipeline = NULL;
    level->dev_allow_pipeline = NULL;

    free(level->host_allow_pipeline);
    level->old_block_amount = 0;
}
