#include "hip/hip_runtime.h"
#include <fightable/level.h>
#include <fightable/tilemap.h>
#include <chrono>

__device__ unsigned char rlCheckCollisionPointRec(Vector2 point, Rectangle rec) {
    if ((point.x >= rec.x) && (point.x < (rec.x + rec.width)) && (point.y >= rec.y) && (point.y < (rec.y + rec.height))) {
        return 1;
    }

    return 0;
}

__global__ void _fLevelOptimizeLevelRenderingCuda(Rectangle area, Vector2 *blocks, unsigned char tile_w,unsigned char tile_h, int block_amount, unsigned char *allows) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= block_amount) return;

    Vector2 block = blocks[i];
    allows[i] = rlCheckCollisionPointRec(Vector2{block.x * tile_w, block.y * tile_h}, area);
}

void _fLevelPrepareCudaRender(struct flevel *level, Rectangle area) {
    int n = level->data_size;

    // auto start_time = std::chrono::high_resolution_clock::now();

    constexpr int threadsPerBlock = 128;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    _fLevelOptimizeLevelRenderingCuda<<<blocksPerGrid, threadsPerBlock>>>(area, level->dev_block_pipeline, level->tilemap->tile_size.x, level->tilemap->tile_size.y, n, level->dev_allow_pipeline);

    hipMemcpy(level->host_allow_pipeline, level->dev_allow_pipeline, n, hipMemcpyDeviceToHost);

    // auto end_time = std::chrono::high_resolution_clock::now();
    // auto time = end_time - start_time;

    // double time_ms = (double)time.count() / (double)1000000;
}