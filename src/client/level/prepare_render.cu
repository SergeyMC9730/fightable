#include "hip/hip_runtime.h"
#include <fightable/level.h>
#include <fightable/tilemap.h>
#include <fightable/block.h>
#include <fightable/state.h>
#include <chrono>
#include <cmath>

__device__ unsigned char rlCheckCollisionPointRec(Vector2 point, Rectangle rec) {
    if ((point.x >= rec.x) && (point.x < (rec.x + rec.width)) && (point.y >= rec.y) && (point.y < (rec.y + rec.height))) {
        return 1;
    }

    return 0;
}

__global__ void _fLevelOptimizeLevelRenderingCuda(Rectangle area, fblock *blocks, unsigned char tile_w,unsigned char tile_h, int block_amount, unsigned char *allows) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= block_amount) return;

    fblock block = blocks[i];
    // check if block id is 0
    if (block.base.tile_x == 40 && block.base.tile_y == 7) return;

    allows[i] = rlCheckCollisionPointRec(Vector2{(float)block.base.block_x * tile_w, (float)block.base.block_y * tile_h}, area);
}

void _fLevelPrepareCudaRender(struct flevel *level, Rectangle area) {
    // hipDeviceSynchronize();

    int n = level->data_size;
    static constexpr float entries = 1.f;

    auto start_time = std::chrono::high_resolution_clock::now();

    int blocks = (int)std::ceil((double)n / entries);

    _fLevelOptimizeLevelRenderingCuda<<<blocks, (int)entries>>>(area, level->dev_block_pipeline, level->tilemap->tile_size.x, level->tilemap->tile_size.y, n, level->dev_allow_pipeline);

    hipMemcpy(level->host_allow_pipeline, level->dev_allow_pipeline, n, hipMemcpyDeviceToHost);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto time = end_time - start_time;

    __state.cuda_time = (double)time.count() / (double)1000000;
}